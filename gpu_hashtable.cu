#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>

#include "gpu_hashtable.hpp"

__device__ int getHash(int data, int limit) {
	return ((long long) abs(data) * 653267llu) % 3452434812973llu % limit;
}

__global__ void kernel_insert(int *keys, int *values, int numEntries, hash_table hashmap) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numEntries)
		return;

	int oldKey, newKey;
	newKey = keys[idx];
	int hash = getHash(newKey, hashmap.size);

	int rangeBegin[2] = {hash, 0};
	int rangeEnd[2] = {hashmap.size, hash};

	for (int r = 0; r <= 1; r++)
		for (int i = rangeBegin[r]; i < rangeEnd[r]; i++) {
			oldKey = atomicCAS(&hashmap.map[0][i].key, KEY_INVALID, newKey);

			if (oldKey == KEY_INVALID || oldKey == newKey) {
				// the position was free
				// only the current thread can enter here because this slot was acquired atomically
				// by the current thread (if oldKey == KEY_INVALID) or the slot was already
				// containing newKey (and no other thread can try to insert this key)
				hashmap.map[0][i].value = values[idx];

				if (oldKey == newKey)
					printf("*");
				return;
			} else {
				oldKey = atomicCAS(&hashmap.map[1][i].key, KEY_INVALID, newKey);

				if (oldKey == KEY_INVALID || oldKey == newKey) {
					hashmap.map[1][i].value = values[idx];

					if (oldKey == newKey)
						printf("*");
					return;
				}
			}
		}
}

__global__ void kernel_get(int *keys, int *values, int numEntries, hash_table hashmap) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numEntries)
		return;

	int key = keys[idx];
	int hash = getHash(keys[idx], hashmap.size);

	int rangeBegin[2] = {hash, 0};
	int rangeEnd[2] = {hashmap.size, hash};

	for (int r = 0; r <= 1; r++) {
		for (int i = rangeBegin[r]; i < rangeEnd[r]; i++) {
			if (hashmap.map[0][i].key == key) {
				values[idx] = hashmap.map[0][i].value;
				return;
			} else if (hashmap.map[1][i].key == key) {
				values[idx] = hashmap.map[1][i].value;
				return;
			}
		}
	}
}

__global__ void kernel_rehash(hash_table oldHash, hash_table newHash) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= oldHash.size)
		return;

	for (int slot = 0; slot <= 1; slot++) {
		if (oldHash.map[slot][idx].key == KEY_INVALID)
			// not a pair here
			continue;

		int oldKey, newKey;
		newKey = oldHash.map[slot][idx].key;
		int hash = getHash(newKey, newHash.size);

		int rangeBegin[2] = {hash, 0};
		int rangeEnd[2] = {newHash.size, hash};

		bool inserted = false;
		for (int r = 0; r <= 1 && !inserted; r++)
			for (int i = rangeBegin[r]; i < rangeEnd[r]; i++) {
				oldKey = atomicCAS(&newHash.map[0][i].key, KEY_INVALID, newKey);

				if (oldKey == KEY_INVALID) {
					newHash.map[0][i].value = oldHash.map[slot][idx].value;
					inserted = true;
					break;
				} else {
					oldKey = atomicCAS(&newHash.map[1][i].key, KEY_INVALID, newKey);

					if (oldKey == KEY_INVALID) {
						newHash.map[1][i].value = oldHash.map[slot][idx].value;
						inserted = true;
						break;
					}
				}
			}
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
//	size = 1000000;
	numInsertedPairs = 0;
	hashmap.size = size;
	hashmap.map[0] = nullptr;
	hashmap.map[1] = nullptr;

	// allocate memory for the new hash map
	for (int slot = 0; slot <= 1; slot++) {
		if (hipMalloc(&hashmap.map[slot], size * sizeof(entry)) != hipSuccess) {
			std::cerr << "Memory allocation error\n";
			return;
		}
		hipMemset(hashmap.map[slot], 0, size * sizeof(entry));
	}
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashmap.map[0]);
	hipFree(hashmap.map[1]);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	hash_table newHashmap;
	newHashmap.size = numBucketsReshape;

	for (int slot = 0; slot <= 1; slot++) {
		if (hipMalloc(&newHashmap.map[slot], numBucketsReshape * sizeof(entry)) != hipSuccess) {
			std::cerr << "Memory allocation error in reshape\n";
			return;
		}
		hipMemset(newHashmap.map[slot], 0, numBucketsReshape * sizeof(entry));
	}

	// load kernel for rehashing all elements from hashmap
	unsigned int numBlocks = hashmap.size / THREADS_PER_BLOCK;
	if (hashmap.size % THREADS_PER_BLOCK != 0) numBlocks++;
	kernel_rehash<<< numBlocks, THREADS_PER_BLOCK >>>(hashmap, newHashmap);

	hipDeviceSynchronize();

	// free old maps' memory and set pointers to the new maps
	for (int slot = 0; slot <= 1; slot++)
		hipFree(hashmap.map[slot]);
	hashmap = newHashmap;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {
	int *deviceKeys, *deviceValues;

	size_t memSize = numKeys * sizeof(int);
	hipMalloc(&deviceKeys, memSize);
	hipMalloc(&deviceValues, memSize);

	if (!deviceKeys || !deviceValues) {
		std::cerr << "Memory allocation error\n";
		return false;
	}

	// check if we need to increase the hashtable's size to reduce the load factor
	if (float(numInsertedPairs + numKeys) / hashmap.size >= MAX_LOAD_FACTOR)
		reshape(int((numInsertedPairs + numKeys) / MIN_LOAD_FACTOR));

	// load keys and values into VRAM
	hipMemcpy(deviceKeys, keys, memSize, hipMemcpyHostToDevice);
	hipMemcpy(deviceValues, values, memSize, hipMemcpyHostToDevice);

	// load kernel for inserting pairs into hashtable
	unsigned int numBlocks = numKeys / THREADS_PER_BLOCK;
	if (numKeys % THREADS_PER_BLOCK != 0) numBlocks++;
	kernel_insert<<< numBlocks, THREADS_PER_BLOCK >>>(deviceKeys,
	                                                  deviceValues, numKeys,
	                                                  hashmap);

	// wait for all insertions to finish
	hipDeviceSynchronize();

	numInsertedPairs += numKeys;

	// free device memory
	hipFree(deviceKeys);
	hipFree(deviceValues);

	return true;
}

/* GET BATCH
 */
int *GpuHashTable::getBatch(int *keys, int numKeys) {
	int *deviceKeys, *values;

	size_t memSize = numKeys * sizeof(int);
	hipMalloc(&deviceKeys, memSize);
	hipMallocManaged(&values, memSize);

	if (!deviceKeys || !values) {
		std::cerr << "Memory allocation error\n";
		return nullptr;
	}

	// load keys and values into VRAM
	hipMemcpy(deviceKeys, keys, memSize, hipMemcpyHostToDevice);

	unsigned int numBlocks = numKeys / THREADS_PER_BLOCK;
	if (numKeys % THREADS_PER_BLOCK != 0) numBlocks++;
	kernel_get<<< numBlocks, THREADS_PER_BLOCK >>>(deviceKeys,
	                                               values, numKeys,
	                                               hashmap);

	hipDeviceSynchronize();

	// free device memory
	hipFree(deviceKeys);

	return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	if (hashmap.size == 0)
		return 0;
	return float(numInsertedPairs) / hashmap.size;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
